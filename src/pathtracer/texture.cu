#include "hip/hip_runtime.h"
#include "texture.h"

namespace CGL {
    DEVICE Vector4D CudaTexture::sample(const Vector2D &uv) {
        // wrap or clamp your UVs as needed
        float u_f = uv.x - floorf(uv.x);
        float v_f = uv.y - floorf(uv.y);
        // // flip v if your image origin is top-left:
        v_f = 1.0f - v_f;

        int u = int(u_f * (width  - 1) + 0.5f);
        int v = int(v_f * (height - 1) + 0.5f);

        // clamp to valid
        u = max(0, min(u, width  - 1));
        v = max(0, min(v, height - 1));

        // compute byte index
        int comps = has_alpha ? 4 : 3;
        size_t idx = (size_t(v) * width + size_t(u)) * comps;
        const uint8_t *base = data;

        uchar4 c;
        if (has_alpha) {
        // RGBA8
        c.x = base[idx + 0];
        c.y = base[idx + 1];
        c.z = base[idx + 2];
        c.w = base[idx + 3];
        } else {
        // RGB8 → treat alpha = 255
        c.x = base[idx + 0];
        c.y = base[idx + 1];
        c.z = base[idx + 2];
        c.w = 255;
        }
        return Vector4D(c.x, c.y, c.z, c.w) * RGB_R;
    }

    DEVICE Vector3D CudaTexture::perturb(const Vector2D &uv, const Vector4D &tangent, const Vector3D &n) {
        Vector4D c = sample(uv);
        Vector3D n_tangent = Vector3D (c.x, c.y, c.z) * 2.0f - Vector3D(1.0f);

        Vector3D T = Vector3D(tangent.x, tangent.y, tangent.z).unit();
        Vector3D B = tangent.w * cross(n, T).unit();

        Vector3D perturbed =
                    (T * n_tangent.x +
                     B * n_tangent.y +
                     n * n_tangent.z).unit();

        printf ("normal: %f %f %f, perturbed: %f %f %f\n", n.x, n.y, n.z, perturbed.x, perturbed.y, perturbed.z);

        return perturbed;
    }
}