#include "hip/hip_runtime.h"
#include "pathtracer/camera.h"
#include "pathtracer/pathtracer.h"
#include "raytraced_renderer.h"

#include <cstddef>
#include <hip/hip_runtime.h>

#include "scene/light.h"

using namespace CGL::SceneObjects;

namespace CGL {

__global__ void kernel_raytrace_temporal(PathTracer* pt) {
    assert (pt != nullptr);
    uint16_t x = ::blockIdx.x * ::blockDim.x + ::threadIdx.x;
    uint16_t y = ::blockIdx.y * ::blockDim.y + ::threadIdx.y;
    
    pt->raytrace_pixel(x,y);
    pt->temporal_resampling(x,y);
}

__global__ void kernel_spatial_sample(PathTracer* pt) {
    assert (pt != nullptr);
    uint16_t x = ::blockIdx.x * ::blockDim.x + ::threadIdx.x;
    uint16_t y = ::blockIdx.y * ::blockDim.y + ::threadIdx.y;
    
    pt->spatial_resampling(x,y);
    pt->render_final_sample(x,y);
}

void RaytracedRenderer::gpu_raytrace() {
    uint16_t width = frameBuffer.w;
    uint16_t height = frameBuffer.h;

    std::cout << "Raytracing on GPU..." << std::endl;


    dim3 blockDim(16, 16);
    dim3 gridDim(
        (width + blockDim.x - 1) / blockDim.x,
        (height + blockDim.y - 1) / blockDim.y
    );
    
    std::cout << "Frame size: " << width << " x " << height << std::endl;
    std::cout << "BlockDim: " << blockDim.x << " x " << blockDim.y << std::endl;
    std::cout << "GridDim: " << gridDim.x << " x " << gridDim.y << std::endl;

    // hipDeviceSetLimit(hipLimitStackSize, 8192);

    std::chrono::time_point<std::chrono::steady_clock> t0 = std::chrono::steady_clock::now();


    kernel_raytrace_temporal<<<gridDim, blockDim>>>(pt_cuda);
    CUDA_ERR(hipGetLastError());
    CUDA_ERR(hipDeviceSynchronize());
    kernel_spatial_sample<<<gridDim, blockDim>>>(pt_cuda);
    CUDA_ERR(hipGetLastError());
    CUDA_ERR(hipDeviceSynchronize());

    std::chrono::time_point<std::chrono::steady_clock> t1 = std::chrono::steady_clock::now();

    std::cout << "Raytracing on GPU done!" << std::endl;
    std::cout << "Time: " << (std::chrono::duration<float>(t1 - t0)).count() << " sec" << std::endl;
    
    CUDA_ERR(hipMemcpy(pt, pt_cuda, sizeof(PathTracer), hipMemcpyDeviceToHost));
    
    auto data_tmp = pt->sampleBuffer.data;
    pt->sampleBuffer.data = (Vector3D*) malloc(width * height * sizeof(Vector3D));
    CUDA_ERR(hipMemcpy(pt->sampleBuffer.data, data_tmp, width * height * sizeof(Vector3D), hipMemcpyDeviceToHost));
    
    // write_to_framebuffer
    pt->sampleBuffer.toColor(frameBuffer, 0, 0, frameBuffer.w, frameBuffer.h);
    free (pt->sampleBuffer.data);

    // restore back
    pt->sampleBuffer.data = data_tmp;
}

void RaytracedRenderer::update_camera(){
    hipMemcpy(pt_cuda, pt, sizeof(PathTracer), hipMemcpyHostToDevice);
    CUDA_ERR(hipGetLastError());
    CUDA_ERR(hipDeviceSynchronize());
}

void RaytracedRenderer::build_accel(std::vector<CudaPrimitive> &primitives, 
                                    std::vector<Vector3D> &vertices,
                                    std::vector<Vector3D> &normals, 
                                    std::vector<Vector2D> &texcoords,
                                    std::vector<Vector4D> &tangents) {
  // build BVH //
  fprintf(stdout, "[PathTracer] Building BVH from %lu primitives... ", primitives.size()); 
  fflush(stdout);
  std::chrono::time_point<std::chrono::steady_clock> t0 = std::chrono::steady_clock::now();

  bvh_cuda = new BVHCuda(primitives, vertices, normals, texcoords, tangents);
  std::chrono::time_point<std::chrono::steady_clock> t1 = std::chrono::steady_clock::now();
  fprintf(stdout, "Done! (%.4f sec)\n", (std::chrono::duration<float>(t1 - t0)).count());
}

void RaytracedRenderer::copy_host_device_pt(std::vector<CudaLight> &lights, std::vector<CudaBSDF> &bsdfs, std::vector<CudaTexture> &textures) {
    std::cout << "Copying PathTracer to GPU..." << std::endl;
    std::cout << "BSDFs size: " << bsdfs.size() << std::endl;
    std::cout << "Lights size: " << lights.size() << std::endl;
    std::cout << "Textures size: " << textures.size() << std::endl;

    //lights
    hipMalloc(&pt->lights, lights.size() * sizeof(CudaLight));
    hipMemcpy(pt->lights, lights.data(), lights.size() * sizeof(CudaLight), hipMemcpyHostToDevice);
    pt->num_lights = lights.size();

    //bsdfs
    hipMalloc(&pt->bsdfs, bsdfs.size() * sizeof(CudaBSDF));
    hipMemcpy(pt->bsdfs, bsdfs.data(), bsdfs.size() * sizeof(CudaBSDF), hipMemcpyHostToDevice);

    //textures
    CudaTexture *textures_cuda = (CudaTexture*) malloc(textures.size() * sizeof(CudaTexture));
    for (size_t i = 0; i < textures.size(); i++) {
        textures_cuda[i].has_alpha = textures[i].has_alpha;
        textures_cuda[i].width = textures[i].width;
        textures_cuda[i].height = textures[i].height;
        int channels = textures[i].has_alpha ? 4 : 3;
        hipMalloc(&textures_cuda[i].data, textures[i].width * textures[i].height * channels);
        hipMemcpy(textures_cuda[i].data, textures[i].data, textures[i].width * textures[i].height * channels, hipMemcpyHostToDevice);
    }
    hipMalloc(&pt->textures, textures.size() * sizeof(CudaTexture));
    hipMemcpy(pt->textures, textures_cuda, textures.size() * sizeof(CudaTexture), hipMemcpyHostToDevice);
    free(textures_cuda);

    //bvh
    hipMalloc(&pt->bvh, sizeof(BVHCuda));
    hipMemcpy(pt->bvh, bvh_cuda, sizeof(BVHCuda), hipMemcpyHostToDevice);

    hipMalloc(&pt->sampleBuffer.data, frameBuffer.w * frameBuffer.h * sizeof(Vector3D));
    
    hipMalloc(&pt->initialSampleBuffer, sizeof(Sample) * frameBuffer.w * frameBuffer.h);
    hipMalloc(&pt->temporalReservoirBuffer, sizeof(Reservoir) * frameBuffer.w * frameBuffer.h);
    hipMalloc(&pt->spatialReservoirBuffer, sizeof(Reservoir) * frameBuffer.w * frameBuffer.h);

    hipMalloc(&pt->rand_states, sizeof(RNGState) * frameBuffer.w * frameBuffer.h);

    PathTracer *pt_cuda;
    hipMalloc(&pt_cuda, sizeof(PathTracer));
    hipMemcpy(pt_cuda, pt, sizeof(PathTracer), hipMemcpyHostToDevice);


    this->pt_cuda = pt_cuda;    
}
}