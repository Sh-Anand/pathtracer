#include "hip/hip_runtime.h"
#include "pathtracer.h"

using namespace CGL::SceneObjects;
using namespace CGL;

namespace CGL {

///< random state for each thread

DEVICE __inline__ void cosine_weighted_hemisphere_sample_3d(RNGState &rand_state, Vector3D *wi, double *pdf) {
  double Xi1 = next_double(rand_state);
  double Xi2 = next_double(rand_state);

  double r = sqrt(Xi1);
  double theta = 2. * PI * Xi2;
  *pdf = sqrt(1-Xi1) / PI;
  *wi = Vector3D(r*cos(theta), r*sin(theta), sqrt(1-Xi1));
}

DEVICE __inline__ Vector3D PathTracer::get_emission(const CudaIntersection &isect) {
  CudaBSDF &bsdf = bsdfs[isect.bsdf_idx];
  Vector2D uv = isect.uv;
  Vector3D emission = bsdf.emissiveFactor * bsdf.emissiveStrength;
  if (bsdf.emission_idx >= 0) {
    Vector4D tc = textures[bsdf.emission_idx].sample(uv);
    emission.x *= tc.x * tc.w;
    emission.y *= tc.y * tc.w;
    emission.z *= tc.z * tc.w;
  }
  return emission;
}

DEVICE __inline__ void PathTracer::perturb_normal(CudaIntersection &isect) {
  int normal_idx = bsdfs[isect.bsdf_idx].normal_idx;
  if (normal_idx < 0) return;

  Vector3D N = isect.n;
  Vector3D T = Vector3D(isect.tangent.x,
                        isect.tangent.y,
                        isect.tangent.z);
  T = (T - N * dot(N, T)).unit();
  Vector3D B = cross(N, T) * isect.tangent.w;

  Vector4D c = textures[normal_idx].sample(isect.uv);
  Vector3D n_tangent = Vector3D(c.x, c.y, c.z) * 2.0f - Vector3D(1.0f);

  Vector3D perturbed = (T * n_tangent.x +
                        B * n_tangent.y +
                        N * n_tangent.z).unit();

  Vector3D diff = perturbed - N;
  double diff_len = diff.norm();
  // use original if diff small to prevent flickering. TODO: better fix
  if (diff_len < 0.4) {
    isect.n = N;
  } else {
    isect.n = perturbed;
  }
}

// following code adapted from https://registry.khronos.org/glTF/specs/2.0/glTF-2.0.html#appendix-b-brdf-implementation
// wo = V, wi = L
DEVICE __inline__ Vector3D PathTracer::f(const CudaIntersection &isect, const Vector3D &wo, const Vector3D &wi, double *occlusion) {
  CudaBSDF &bsdf = bsdfs[isect.bsdf_idx];
  Vector3D N = isect.n; // perturbed normal
  Vector2D uv = isect.uv;

  // 1) geometry terms
  Vector3D H = (wo + wi).unit(); // bisector
  double NoV   = fabs(dot(N, wo));
  double NoL   = fabs(dot(N, wi));
  if (NoL == 0 || NoV == 0) return Vector3D(0.0);
  double NoH   = dot(N, H);
  double VoH   = dot(wo, H);
  double LoH   = dot(wi, H);

  // 2) get base texture
  Vector3D base = Vector3D(bsdf.baseColor.x,
                           bsdf.baseColor.y,
                           bsdf.baseColor.z);
  if (bsdf.tex_idx >= 0) {
    Vector4D t = textures[bsdf.tex_idx].sample(uv);
    base = base * Vector3D(t.x, t.y, t.z);
  }

  // 3) get metallic roughness
  double metal    = bsdf.metallic;
  double roughness= bsdf.roughness;
  if (bsdf.orm_idx >= 0) {
    Vector4D orm = textures[bsdf.orm_idx].sample(uv);
    metal     = orm.z;
    roughness = orm.y;
    *occlusion = orm.x;
  }

  // clamp values : safety
  metal     = clamp_device(metal,     0.0, 1.0);
  roughness = clamp_device(roughness, 0.04, 1.0); // avoid ->0
  double onemmetal = 1.0 - metal;

  double alpha = roughness * roughness;

  // 4) diffuse and specular components
  Vector3D c_diff = base * onemmetal;
  Vector3D f0 = onemmetal * Vector3D(0.04) + metal * base;
  Vector3D F = f0 + (Vector3D(1.0) - f0) * pow(1.0 - VoH, 5.0);
  Vector3D f_diffuse = (Vector3D(1.0) - F) * PI_R * c_diff;

  double D = D_compute(alpha, NoH);
  double V = G_compute(alpha, NoV, NoL, VoH, LoH) / (4.0 * NoV * NoL);
  Vector3D f_specular = F * D * V;

  return f_diffuse + f_specular;
}

// Importance‑sample both diffuse (Lambert) and GGX specular lobes of the metallic‑roughness BRDF.
// Returns f(wo, *wi), writes out *wi, *pdf, and *occlusion.
DEVICE __inline__ Vector3D PathTracer::sample_f(const CudaIntersection &isect,
                                                const Vector3D       &wo,
                                                Vector3D             *wi,
                                                double               *pdf,
                                                double               *occlusion,
                                                RNGState             &rand_state) {
  // 1) Material & normal
  const CudaBSDF &bsdf = bsdfs[isect.bsdf_idx];
  Vector3D N    = isect.n;
  Vector2D uv   = isect.uv;
  *occlusion    = 1.0;

  // 2) Base color
  Vector3D base = Vector3D(bsdf.baseColor.x,
                           bsdf.baseColor.y,
                           bsdf.baseColor.z);
  if (bsdf.tex_idx >= 0) {
    Vector4D t = textures[bsdf.tex_idx].sample(uv);
    base = base * Vector3D(t.x, t.y, t.z);
  }

  // 3) Metallic, roughness, occlusion from ORM
  double metal     = clamp_device(bsdf.metallic,  0.0, 1.0);
  double roughness = clamp_device(bsdf.roughness, 0.02,1.0);
  if (bsdf.orm_idx >= 0) {
    Vector4D orm = textures[bsdf.orm_idx].sample(uv);
    *occlusion   = orm.x;
    roughness    = orm.y;
    metal        = orm.z;
  }
  double onem = 1.0 - metal;

  // // 4) Visibility check
  double NoV = fabs(dot(N, wo));
  if (NoV == 0.0) {
    *pdf = 0.0;
    return Vector3D(0.0);
  }

  // 5) Precompute F₀ and mixture weights
  double alpha = roughness * roughness;
  Vector3D F0  = Vector3D(0.04) * onem + base * metal;

  double P_d = onem;  // diffuse weight
  double P_s = metal; // specular weight
  double w   = P_d + P_s;
  P_d /= w;
  P_s /= w;

  // 6) Randomly choose lobe
  double u = next_double(rand_state);
  if (u < P_d) {
    // ── DIFFUSE ──
    // sample cosine‑weighted hemisphere
    cosine_weighted_hemisphere_sample_3d(rand_state, wi, pdf);
    *pdf *= P_d;

    // evaluate BRDF
    Vector3D H     = (wo + *wi).unit();
    double VoH     = max(dot(wo, H), 0.0);
    Vector3D F_geo = F0 + (Vector3D(1.0) - F0) * pow(1.0 - VoH, 5.0);
    Vector3D c_diff = base * onem;
    return (Vector3D(1.0) - F_geo) * (1.0 / M_PI) * c_diff;
  } else {
    // ── SPECULAR (GGX) ──
    // (a) sample microfacet normal H via GGX NDF
    double r1 = next_double(rand_state);
    double r2 = next_double(rand_state);
    double phi      = 2.0 * M_PI * r1;
    double cosTheta = sqrt((1.0 - r2) / (1.0 + (alpha*alpha - 1.0) * r2));
    double sinTheta = sqrt(max(0.0, 1.0 - cosTheta*cosTheta));

    Matrix3x3 o2w;
    make_coord_space(o2w, N);
    Vector3D localH = Vector3D(sinTheta * cos(phi),
                               sinTheta * sin(phi),
                               cosTheta);
    Vector3D H = (o2w * localH).unit();

    // (b) reflect view vector about H
    *wi = reflect(-wo, H);

    // (c) compute PDF
    double NoH   = max(dot(N, H), 0.0);
    double VoH   = max(dot(wo, H), 0.0);
    double D     = D_compute(alpha, NoH);
    double pdf_H = D * NoH;
    double pdf_w = pdf_H / (4.0 * VoH);
    *pdf = pdf_w * P_s;

    // (d) evaluate microfacet BRDF
    double NoL = max(dot(N, *wi), 0.0);
    double G   = G_compute(alpha, NoV, NoL, VoH, max(dot(*wi, H), 0.0));
    Vector3D F_geo = F0 + (Vector3D(1.0) - F0) * pow(1.0 - VoH, 5.0);
    return F_geo * (D * G / (4.0 * NoV * NoL));
  }
}

// power­-heuristic MIS weight, β=2
inline __device__ double mis_weight(double pA, double pB) {
  double wA = pA*pA;
  double wB = pB*pB;
  return wA / (wA + wB);
}

// mixture PDF of your metallic‑roughness lobes
DEVICE __inline__ double PathTracer::bsdf_pdf(const CudaIntersection &isect,
                                  const Vector3D &wo,
                                  const Vector3D &wi) {
  Vector3D N = isect.n;
  double NoL = fabs(dot(N, wi));
  if (NoL == 0) return 0.0;

  // fetch metallic & roughness
  CudaBSDF &b = bsdfs[isect.bsdf_idx];
  double metal    = clamp_device(b.metallic,  0.0, 1.0);
  double roughness= clamp_device(b.roughness, 0.02,1.0);
  double onem     = 1.0 - metal;
  double alpha    = roughness * roughness;

  // 1) diffuse pdf = (cosθ/π)
  double pdf_diff = onem * (NoL / M_PI);

  // 2) specular pdf = D(α,NoH)·NoH / (4·VoH)
  Vector3D H   = (wo + wi).unit();
  double NoH   = fmax(dot(N, H), 0.0);
  double VoH   = fmax(dot(wo, H), 0.0);
  double D     = D_compute(alpha, NoH);
  double pdf_spec = metal * (D * NoH / (4.0 * VoH));

  return pdf_diff + pdf_spec;
}


DEVICE Vector3D PathTracer::estimate_direct_lighting_importance(Ray &r,
                                                const CudaIntersection &isect) {
  Matrix3x3 o2w;

  make_coord_space(o2w, isect.n);
  Matrix3x3 w2o = o2w.T();

  // w_out points towards the source of the ray (e.g.,
  // toward the camera if this is a primary ray)
  const Vector3D hit_p = r.o + r.d * isect.t;
  const Vector3D w_out = w2o * (-r.d);
  Vector3D L_out = Vector3D(0, 0, 0);
  Vector3D wi;

  //NOTE: wi here is in worldpsace,

  uint16_t x = blockIdx.x * blockDim.x + threadIdx.x;
  uint16_t y = blockIdx.y * blockDim.y + threadIdx.y;

  double occlusion; //ignored for dir lighting
  for (int i = 0; i < num_lights; ++i) {
    CudaLight &L = lights[i];
    Vector3D wi;
    double   distToL, pdfL;
    Vector3D Li = L.sample_L(hit_p, &wi, &distToL, &pdfL,
                           rand_states[x + y * sampleBuffer.w], bvh->vertices);

    double cosNL = fmax(dot(isect.n, wi), 0.0);
    if (pdfL > 0 && cosNL > 0) {
      // shadow test
      Ray shadow(hit_p, wi);
      shadow.min_t = EPS_D;
      shadow.max_t = distToL;
      if (!bvh->has_intersect(shadow)) {
        // BRDF eval and PDF of sampling that same wi via BSDF
        Vector3D f_val = f(isect, w_out, wi, &occlusion);
        double  pdfB   = bsdf_pdf(isect, w_out, wi);
        double  w      = mis_weight(pdfL, pdfB);
        L_out += f_val * Li * cosNL * w / pdfL;
      }
    }
  }

  Vector3D wi_bsdf;
  double   pdfB;
  Vector3D f_bsdf = sample_f(isect, w_out, &wi_bsdf, &pdfB,
                             &occlusion, rand_states[x + y * sampleBuffer.w]);
  double cosNL = fmax(dot(isect.n, wi_bsdf), 0.0);

  if (pdfB > 0 && cosNL > 0) {
    // trace a ray in that direction and see if it hits *any* light
    Ray shadow(hit_p, wi_bsdf);
    CudaIntersection Lhit;
    shadow.min_t = EPS_D;
    shadow.max_t = INFINITY;

    // bad, checking every light, assumes few lights
    CudaBSDF &bsdf = bsdfs[isect.bsdf_idx];

    for (int i = 0; i < num_lights; ++i) {
      CudaLight &L = lights[i];
      double pdfL;
      if (L.has_intersect(shadow, hit_p, isect.n, bvh->vertices, &pdfL)) {
        // get the light and compute its PDF for this direction
        Vector3D Li = bsdf.emissiveFactor * bsdf.emissiveStrength;
        double w    = mis_weight(pdfB, pdfL);
        L_out += f_bsdf * Li * cosNL * w / pdfB;
      }
    }
    
  }

  return L_out;
}


#define RRT 0.7f

DEVICE Vector3D PathTracer::at_least_one_bounce_radiance(Ray& r, const CudaIntersection& isect_init) {
    Vector3D L_out_total(0.0);
    Vector3D throughput(1.0);
    Ray current_ray = r;
    CudaIntersection isect = isect_init;
    bool first_bounce = true;

    // constant index since x,y don’t change across bounces
    int idx = current_ray.x + current_ray.y * sampleBuffer.w;

    while (true) {
        // build shading frame
        Matrix3x3 o2w;
        make_coord_space(o2w, isect.n);
        Matrix3x3 w2o = o2w.T();

        // hit point & outgoing dir in local space
        Vector3D hit_p  = current_ray.o + current_ray.d * isect.t;
        Vector3D w_out  = w2o * (-current_ray.d);

        // direct lighting
        Vector3D L_out = estimate_direct_lighting_importance(current_ray, isect);
        if (first_bounce) {
            initialSampleBuffer[idx].emittance = L_out;
        }
        L_out_total += throughput * L_out;

        // russian-roulette survival
        float p_survive = (current_ray.depth == 1) ? 1.0f : RRT;
        if (current_ray.depth > 1 &&
            next_double(rand_states[idx]) >= RRT)
            break;

        // sample BSDF
        Vector3D wi;
        double pdf;
        double occlusion = 1.0;
        Vector3D fcos = occlusion * sample_f(isect, w_out, &wi, &pdf, &occlusion, rand_states[idx]) * abs_cos_theta(wi);
        if (pdf <= 0.0)
            break;

        // update throughput
        throughput = throughput * (first_bounce ? 1.0 : fcos);
        throughput /= (pdf * p_survive);

        // spawn next ray
        Ray bounce_ray(hit_p, o2w * wi);
        bounce_ray.min_t = EPS_D;
        bounce_ray.depth = current_ray.depth + 1;
        bounce_ray.x = current_ray.x;
        bounce_ray.y = current_ray.y;

        CudaIntersection bounce_isect;
        if (!bvh->intersect(bounce_ray, &bounce_isect))
            break;

        // perturb_normal(bounce_isect);

        if (first_bounce) {
            Vector3D bounce_p = bounce_ray.o + bounce_ray.d * bounce_isect.t;
            Sample* s = &initialSampleBuffer[idx];
            s->x_v   = hit_p;
            s->n_v   = isect.n;
            s->x_s   = bounce_p;
            s->n_s   = bounce_isect.n;
            s->pdf   = pdf;
            s->fcos  = fcos;
        }

        // prepare for next iteration
        current_ray = bounce_ray;
        isect       = bounce_isect;
        first_bounce = false;
    }

    return L_out_total - initialSampleBuffer[idx].emittance;
}

DEVICE Vector3D PathTracer::est_radiance_global_illumination(Ray &r) {
  CudaIntersection isect;
  Vector3D L_out;

  if (!bvh->intersect(r, &isect))
    return L_out;

  L_out = get_emission(isect) + at_least_one_bounce_radiance(r, isect);

  return L_out;
}

DEVICE void PathTracer::raytrace_pixel(uint16_t x, uint16_t y) {
  CudaIntersection isect;
  
  uint16_t num_samples = ns_aa;
  Ray r;
  uint16_t i = 1;
  initialSampleBuffer[x + y * sampleBuffer.w] = Sample();
  init_gpu_rng(rand_states[x + y * sampleBuffer.w], 1234 + x + y * sampleBuffer.w);
  do {
    Vector2D origin = Vector2D(x, y);
    Vector2D sample = origin + Vector2D(next_double(rand_states[x + y * sampleBuffer.w]), next_double(rand_states[x + y * sampleBuffer.w]));
    r = camera.generate_ray(sample.x / sampleBuffer.w, sample.y / sampleBuffer.h);
    r.depth = 1, r.x = x, r.y = y;
  } while (i++ != num_samples && !bvh->intersect(r, &isect));
  if (i == num_samples + 1) {
    initialSampleBuffer[x + y * sampleBuffer.w].L = Vector3D(0, 0, 0);
  } else {
    perturb_normal(isect);
    Vector3D L = at_least_one_bounce_radiance(r, isect);
    initialSampleBuffer[r.x + r.y * sampleBuffer.w].emittance += get_emission(isect);
    initialSampleBuffer[r.x + r.y * sampleBuffer.w].L = L;
  }
}

// Computes jacobian from s1->s2 as defined in Equation 11 of the ReSTIR-GI paper
DEVICE __inline__ double jacobian(const Sample& s1, const Sample& s2) {
    Vector3D xq1 = s1.x_v;
    Vector3D xq2 = s1.x_s;
    Vector3D xr1 = s2.x_v;

    Vector3D nq2 = s1.n_s;

    double cos_phi_q2 = fabs(dot(nq2, (xq1 - xq2).unit())); 
    double cos_phi_r2 = fabs(dot(nq2, (xr1 - xq2).unit()));

    double distance_q = (xq1 - xq2).norm2();
    double distance_r = (xr1 - xq2).norm2();

    return (cos_phi_r2 / cos_phi_q2) * (distance_q / distance_r);
}

DEVICE void PathTracer::temporal_resampling(uint16_t x, uint16_t y) {
  Sample S = initialSampleBuffer[x + y * sampleBuffer.w];
  Reservoir R = Reservoir();

  double w = p_hat(S);
  R.update(S, w, rand_states[x + y * sampleBuffer.w]);
  R.W = R.w / (R.M * p_hat(R.z));

  temporalReservoirBuffer[x + y * sampleBuffer.w] = R;
  spatialReservoirBuffer[x + y * sampleBuffer.w] = R;
}

DEVICE void PathTracer::spatial_resampling(uint16_t x, uint16_t y) {

  const uint16_t neighbouring_pixel_radius = floor(0.1 * min(sampleBuffer.w, sampleBuffer.h));

  Reservoir Rs = spatialReservoirBuffer[x + y * sampleBuffer.w];
  Sample q = initialSampleBuffer[x + y * sampleBuffer.w];
  RNGState rand_state = rand_states[x + y * sampleBuffer.w];
  const uint8_t max_neighbouring_samples = 9; // ReSTIR GI paper value without temporal sampling
  for (uint8_t s = 0; s < max_neighbouring_samples; s++) {
    // Randomly choose a neighbor pixel qn
    int window = 2 * neighbouring_pixel_radius + 1;
    uint16_t sample_x = x + static_cast<int>(next_double(rand_state) * window) - neighbouring_pixel_radius;
    uint16_t sample_y = y + static_cast<int>(next_double(rand_state) * window) - neighbouring_pixel_radius;

    // Ensure the sample is within the frame buffer bounds
    if (sample_x >= sampleBuffer.w || sample_y >= sampleBuffer.h) continue;

    // Retrieve the reservoir from the neighboring pixel
    Reservoir Rn = temporalReservoirBuffer[sample_x + sample_y * sampleBuffer.w];
    // Calculate geometric similarity between q and qn
    if (!are_geometrically_similar(q, Rn.z) || Rn.z.L == Vector3D(0, 0, 0)) continue;

    // Calculate |Jqn→q| (Jacobian determinant)
    double Jqn_to_q = jacobian(Rn.z, q); // Placeholder for actual Jacobian calculation

    // Calculate ˆp′q
    double p_prime_q = (p_hat(Rn.z)) / Jqn_to_q;

    // visibility test
    // if neighbour's path's point is invisible from the current path's point, p_prime_q = 0
    Ray shadow_ray(q.x_v, (Rn.z.x_s - q.x_v).unit());
    shadow_ray.min_t = EPS_D;
    shadow_ray.max_t = (Rn.z.x_s - q.x_v).norm() - EPS_D;
    if (bvh->has_intersect(shadow_ray)) p_prime_q = 0;

    // Merge Rn into the current reservoir
    Rs.merge(Rn, p_prime_q, rand_state);
  }

  double phat = p_hat(Rs.z);
  Rs.W = Rs.M * phat > 0 ? Rs.w / (Rs.M * phat) : 0;
  spatialReservoirBuffer[x + y * sampleBuffer.w] = Rs;
  rand_states[x + y * sampleBuffer.w] = rand_state;
}

DEVICE void PathTracer::render_final_sample(uint16_t x, uint16_t y) {
  Reservoir R = spatialReservoirBuffer[x + y * sampleBuffer.w];
  Sample S = R.z;
  Sample initial = initialSampleBuffer[x + y *  sampleBuffer.w];
  Vector3D L = initial.emittance + S.fcos * S.L * R.W;

  sampleBuffer.update_pixel(L, x, y);
}

}