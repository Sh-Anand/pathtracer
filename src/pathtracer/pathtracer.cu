#include "hip/hip_runtime.h"
#include "pathtracer.h"

using namespace CGL::SceneObjects;
using namespace CGL;

namespace CGL {

///< random state for each thread

DEVICE __inline__ void cosine_weighted_hemisphere_sample_3d(RNGState &rand_state, Vector3D *wi, float *pdf) {
  float Xi1 = next_float(rand_state);
  float Xi2 = next_float(rand_state);

  float r = sqrt(Xi1);
  float theta = 2. * PI * Xi2;
  *pdf = sqrt(1-Xi1) / PI;
  *wi = Vector3D(r*cos(theta), r*sin(theta), sqrt(1-Xi1));
}


DEVICE __inline__ Vector3D uniform_hemisphere_sample_3d(RNGState &rand_state) {
  float z = next_float(rand_state) * 2 - 1;
  float sinTheta = sqrtf(fmaxf(0.0f, 1.0f - z * z));

  float phi = 2.0f * PI_F * next_float(rand_state);

  return Vector3D(cos(phi) * sinTheta, sin(phi) * sinTheta, z);
}

DEVICE __inline__ Vector3D sample_f(const CudaBSDF *bsdf, const Vector3D wo, Vector3D *wi, float *pdf, RNGState &rand_state) {
  cosine_weighted_hemisphere_sample_3d(rand_state, wi, pdf);
  return bsdf->f(wo, *wi);
}

DEVICE __inline__ Vector3D sample_L(const CudaAmbientLight *light, const Vector3D p, Vector3D* wi,
                             float* distToLight, float* pdf, RNGState &rand_state) {
  Vector3D dir = uniform_hemisphere_sample_3d(rand_state);
  *wi = light->sampleToWorld * dir;
  *distToLight = INFINITY;
  *pdf = 1.0 / (2.0 * PI_F);
  return light->radiance;
}

DEVICE __inline__ Vector3D sample_L(const CudaDirectionalLight *light, const Vector3D p, Vector3D* wi,
                                    float* distToLight, float* pdf) {
  *wi = light->dirToLight;
  *distToLight = INFINITY;
  *pdf = 1.0;
  return light->radiance;
}

DEVICE __inline__ Vector3D sample_L(const CudaPointLight *light, const Vector3D p, Vector3D* wi,
                             float* distToLight,
                             float* pdf) {
  Vector3D d = light->position - p;
  *wi = d.unit();
  *distToLight = d.norm();
  *pdf = 1.0;
  return light->radiance;
}

DEVICE __inline__ Vector3D sample_L(const CudaAreaLight *light, const Vector3D p, Vector3D* wi, 
                             float* distToLight, float* pdf, RNGState &rand_state) {
  Vector2D sample = Vector2D(next_float(rand_state), next_float(rand_state)) - Vector2D(0.5f, 0.5f);
  Vector3D d = light->position + sample.x * light->dim_x + sample.y * light->dim_y - p;
  float cosTheta = dot(d, light->direction);
  float sqDist = d.norm2();
  float dist = sqrtf(sqDist);
  *wi = d / dist;
  *distToLight = dist;
  *pdf = sqDist / (light->area * fabs(cosTheta));
  return cosTheta < 0 ? light->radiance : Vector3D();
}

DEVICE __inline__ Vector3D p_sample_L(const CudaLight *light, const Vector3D p,
                             Vector3D* wi, float* distToLight,
                             float* pdf, RNGState &rand_state) {
  switch (light->type) {
    case DIRECTIONAL:
      return sample_L(&light->light.directional, p ,wi, distToLight, pdf);
    case POINT:
      return sample_L(&light->light.point, p, wi, distToLight, pdf);
    case AREA:
      return sample_L(&light->light.area, p, wi, distToLight, pdf, rand_state);
    default:
      return Vector3D(0, 0, 0);
  }
  return Vector3D();
}

DEVICE Vector3D PathTracer::estimate_direct_lighting_importance(Ray &r,
                                                const CudaIntersection &isect) {
  // Estimate the lighting from this intersection coming directly from a light.
  // To implement importance sampling, sample only from lights, not uniformly in
  // a hemisphere.

  // make a coordinate system for a hit point
  // with N aligned with the Z direction.
  Matrix3x3 o2w;
  make_coord_space(o2w, isect.n);
  Matrix3x3 w2o = o2w.T();

  // w_out points towards the source of the ray (e.g.,
  // toward the camera if this is a primary ray)
  const Vector3D hit_p = r.o + r.d * isect.t;
  const Vector3D w_out = w2o * (-r.d);
  Vector3D L_out = Vector3D(0, 0, 0);
  Vector3D wi;
  float distToLight, pdf;

  //NOTE: wi here is in worldpsace, unlike in the previous function
  size_t sample_count = 0;
  for (int i = 0; i < num_lights; i++) {
    const CudaLight *light = &lights[i];
    int num_samples = light->is_delta_light() ? 1 : ns_area_light;
    sample_count += num_samples;
    for (int j = 0; j < num_samples; j++) {
      Vector3D radiance = p_sample_L(light, hit_p, &wi, &distToLight, &pdf, rand_states[r.x + r.y * sampleBuffer.w]);
      Vector3D wi_o = w2o * wi;
      if (wi_o.z < 0 || radiance == 0) continue;
      Ray shadow_ray = Ray(hit_p, wi);
      shadow_ray.min_t = EPS_F;
      shadow_ray.max_t = distToLight;
      CudaIntersection light_isect;
      if (!bvh->intersect(shadow_ray, &light_isect)) {
        L_out += bsdfs[isect.bsdf_idx].f(w_out, wi_o) * radiance * abs_cos_theta(wi_o) / pdf;
      }
    }
  }

  L_out /= sample_count;
  return L_out;
}


#define RRT 0.7

DEVICE Vector3D PathTracer::at_least_one_bounce_radiance(Ray& initial_r,
                                                    const CudaIntersection& initial_isect) {

        Vector3D total_radiance(0.0);
        Vector3D path_throughput(1.0);
        Ray current_ray = initial_r;
        CudaIntersection current_isect = initial_isect;
        int current_depth = initial_r.depth;

        Vector3D first_bounce_hit_p;
        Vector3D first_bounce_n_v;

        while (true) {
            Matrix3x3 o2w;
            make_coord_space(o2w, current_isect.n);
            Matrix3x3 w2o = o2w.T();

            Vector3D hit_p = current_ray.o + current_ray.d * current_isect.t;
            Vector3D w_out_local = w2o * (-current_ray.d);

            Vector3D L_direct_or_emitted = estimate_direct_lighting_importance(current_ray, current_isect);
            total_radiance += path_throughput * L_direct_or_emitted;

            if (current_depth == 1) {
                initialSampleBuffer[initial_r.x + initial_r.y * sampleBuffer.w].emittance = L_direct_or_emitted;
                first_bounce_hit_p = hit_p;
                first_bounce_n_v = current_isect.n;
            }

            bool continue_path = (current_depth == 1) || (current_depth < max_ray_depth && (next_float(rand_states[current_ray.x + current_ray.y * sampleBuffer.w]) < RRT));

            if (!continue_path) {
                break;
            }

            Vector3D wi_local;
            float pdf;
            Vector3D f = sample_f(&bsdfs[current_isect.bsdf_idx], w_out_local, &wi_local, &pdf, rand_states[current_ray.x + current_ray.y * sampleBuffer.w]);

            if (pdf <= 1e-6 || f == Vector3D(0, 0, 0)) {
                break;
            }

            double cos_theta_i = abs_cos_theta(wi_local);

            Vector3D throughput_update = f * cos_theta_i / pdf;

            if (current_depth > 1 && continue_path) {
                 throughput_update /= RRT;
            }
            path_throughput = path_throughput * throughput_update;

            Vector3D bounce_dir_world = o2w * wi_local;
            Ray bounce_ray(hit_p, bounce_dir_world);
            bounce_ray.min_t = EPS_F;
            bounce_ray.depth = current_depth + 1;

            CudaIntersection next_isect;
            bool hit = bvh->intersect(bounce_ray, &next_isect);

            if (current_depth == 1) {
                Sample* s = &initialSampleBuffer[initial_r.x + initial_r.y * sampleBuffer.w];
                s->x_v = first_bounce_hit_p;
                s->n_v = first_bounce_n_v;
                s->pdf = pdf;
                s->fcos = f * cos_theta_i;

                if (hit) {
                    Vector3D bounce_p = bounce_ray.o + bounce_ray.d * next_isect.t;
                    s->x_s = bounce_p;
                    s->n_s = next_isect.n;
                    s->L = path_throughput;
                } else {
                    s->x_s = Vector3D();
                    s->n_s = Vector3D();
                    s->L = Vector3D();
                }
            }

            if (!hit) {
                break;
            }

            current_ray = bounce_ray;
            current_isect = next_isect;
            current_depth++;

        } // End while loop

        return total_radiance;
    }


DEVICE Vector3D PathTracer::est_radiance_global_illumination(Ray &r) {
  CudaIntersection isect;
  Vector3D L_out;

  if (!bvh->intersect(r, &isect))
    return L_out;

  L_out = bsdfs[isect.bsdf_idx].get_emission() + at_least_one_bounce_radiance(r, isect);

  return L_out;
}

DEVICE void PathTracer::raytrace_pixel(uint16_t x, uint16_t y) {
  CudaIntersection isect;
  
  size_t num_samples = ns_aa;
  Ray r;
  size_t i = 1;
  initialSampleBuffer[x + y * sampleBuffer.w] = Sample();
  init_gpu_rng(rand_states[x + y * sampleBuffer.w], 1234 + x + y * sampleBuffer.w);
  do {
    Vector2D origin = Vector2D(x, y);
    Vector2D sample = origin + Vector2D(next_float(rand_states[x + y * sampleBuffer.w]), next_float(rand_states[x + y * sampleBuffer.w]));
    r = camera.generate_ray(sample.x / sampleBuffer.w, sample.y / sampleBuffer.h);
    r.depth = 1, r.x = x, r.y = y;
  } while (i++ != num_samples && !bvh->intersect(r, &isect));
  if (i == num_samples + 1) {
    initialSampleBuffer[x + y * sampleBuffer.w].L = Vector3D(0, 0, 0);
  } else {
    Vector3D L = at_least_one_bounce_radiance(r, isect);
    initialSampleBuffer[r.x + r.y * sampleBuffer.w].emittance += bsdfs[isect.bsdf_idx].get_emission();
  }
}

// Computes jacobian from s1->s2 as defined in Equation 11 of the ReSTIR-GI paper
DEVICE __inline__ float jacobian(const Sample& s1, const Sample& s2) {
    Vector3D xq1 = s1.x_v;
    Vector3D xq2 = s1.x_s;
    Vector3D xr1 = s2.x_v;

    Vector3D nq2 = s1.n_s;

    float cos_phi_q2 = fabsf(dot(nq2, (xq1 - xq2).unit())); 
    float cos_phi_r2 = fabsf(dot(nq2, (xr1 - xq2).unit()));

    float distance_q = (xq1 - xq2).norm2();
    float distance_r = (xr1 - xq2).norm2();

    return (cos_phi_r2 / cos_phi_q2) * (distance_q / distance_r);
}

DEVICE void PathTracer::temporal_resampling(uint16_t x, uint16_t y) {
  Sample S = initialSampleBuffer[x + y * sampleBuffer.w];
  Reservoir R = Reservoir();

  float w = p_hat(S);
  R.update(S, w, rand_states[x + y * sampleBuffer.w]);
  R.W = R.w / (R.M * p_hat(R.z));

  temporalReservoirBuffer[x + y * sampleBuffer.w] = R;
  spatialReservoirBuffer[x + y * sampleBuffer.w] = R;
}

DEVICE void PathTracer::spatial_resampling(uint16_t x, uint16_t y) {

  const uint16_t neighbouring_pixel_radius = floor(0.1 * min(sampleBuffer.w, sampleBuffer.h));

  Reservoir Rs = spatialReservoirBuffer[x + y * sampleBuffer.w];
  Sample q = initialSampleBuffer[x + y * sampleBuffer.w];
  RNGState rand_state = rand_states[x + y * sampleBuffer.w];
  const uint8_t max_neighbouring_samples = 9; // ReSTIR GI paper value without temporal sampling
  for (uint8_t s = 0; s < max_neighbouring_samples; s++) {
    // Randomly choose a neighbor pixel qn
    int window = 2 * neighbouring_pixel_radius + 1;
    uint16_t sample_x = x + static_cast<int>(next_float(rand_state) * window) - neighbouring_pixel_radius;
    uint16_t sample_y = y + static_cast<int>(next_float(rand_state) * window) - neighbouring_pixel_radius;

    // Ensure the sample is within the frame buffer bounds
    if (sample_x >= sampleBuffer.w || sample_y >= sampleBuffer.h) continue;

    // Retrieve the reservoir from the neighboring pixel
    Reservoir Rn = temporalReservoirBuffer[sample_x + sample_y * sampleBuffer.w];
    // Calculate geometric similarity between q and qn
    if (!are_geometrically_similar(q, Rn.z) || Rn.z.L == Vector3D(0, 0, 0)) continue;

    // Calculate |Jqn→q| (Jacobian determinant)
    float Jqn_to_q = jacobian(Rn.z, q); // Placeholder for actual Jacobian calculation

    // Calculate ˆp′q
    float p_prime_q = (p_hat(Rn.z)) / Jqn_to_q;

    // visibility test
    // if neighbour's path's point is invisible from the current path's point, p_prime_q = 0
    Ray shadow_ray(q.x_v, (Rn.z.x_s - q.x_v).unit());
    shadow_ray.min_t = EPS_F;
    shadow_ray.max_t = (Rn.z.x_s - q.x_v).norm() - EPS_F;
    CudaIntersection isect;
    if (bvh->intersect(shadow_ray, &isect)) p_prime_q = 0;

    // Merge Rn into the current reservoir
    Rs.merge(Rn, p_prime_q, rand_state);
  }

  float phat = p_hat(Rs.z);
  Rs.W = Rs.M * phat > 0 ? Rs.w / (Rs.M * phat) : 0;
  spatialReservoirBuffer[x + y * sampleBuffer.w] = Rs;
  rand_states[x + y * sampleBuffer.w] = rand_state;
}

DEVICE void PathTracer::render_final_sample(uint16_t x, uint16_t y) {
  Reservoir R = spatialReservoirBuffer[x + y * sampleBuffer.w];
  Sample S = R.z;
  Sample initial = initialSampleBuffer[x + y *  sampleBuffer.w];
  Vector3D L = initial.emittance + initial.fcos * S.L * R.W;

  sampleBuffer.update_pixel(L, x, y);
}

}