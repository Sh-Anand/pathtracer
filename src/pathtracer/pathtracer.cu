#include "hip/hip_runtime.h"
#include "pathtracer.h"

#include "scene/light.h"
#include "scene/sphere.h"
#include "scene/triangle.h"

#include <hiprand/hiprand_kernel.h> 


using namespace CGL::SceneObjects;
using namespace CGL;

namespace CGL {

/**
 * This function creates a object space (basis vectors) from the normal vector
 */

DEVICE __inline__ void cosine_weighted_hemisphere_sample_3d(hiprandState *rand_state, Vector3D *wi, double *pdf) {
  double Xi1 = hiprand_uniform(rand_state);
  double Xi2 = hiprand_uniform(rand_state);

  double r = sqrt(Xi1);
  double theta = 2. * PI * Xi2;
  *pdf = sqrt(1-Xi1) / PI;
  *wi = Vector3D(r*cos(theta), r*sin(theta), sqrt(1-Xi1));
}

DEVICE __inline__ Vector3D sample_f(CudaDiffuseBSDF *bsdf, const Vector3D wo, Vector3D *wi, double *pdf, hiprandState *rand_state) {
  cosine_weighted_hemisphere_sample_3d(rand_state, wi, pdf);
  return bsdf->f(wo, *wi);
}

DEVICE __inline__ Vector3D sample_f(CudaEmissionBSDF *bsdf, const Vector3D wo, Vector3D *wi, double *pdf, hiprandState *rand_state) {
  *pdf = 1.0 / PI;
  cosine_weighted_hemisphere_sample_3d(rand_state, wi, pdf);
  return Vector3D();
}

DEVICE __inline__ Vector3D sample_L(CudaDirectionalLight *light, const Vector3D p, Vector3D* wi,
                                    double* distToLight, double* pdf) {
  *wi = light->dirToLight;
  *distToLight = INFINITY;
  *pdf = 1.0;
  return light->radiance;
}

DEVICE __inline__ Vector3D sample_L(CudaPointLight *light, const Vector3D p, Vector3D* wi,
                             double* distToLight,
                             double* pdf) {
  Vector3D d = light->position - p;
  *wi = d.unit();
  *distToLight = d.norm();
  *pdf = 1.0;
  return light->radiance;
}

DEVICE __inline__ Vector3D sample_L(CudaAreaLight *light, const Vector3D p, Vector3D* wi, 
                             double* distToLight, double* pdf, hiprandState *rand_state) {
  Vector2D sample = Vector2D(1 - hiprand_uniform(rand_state), 1 - hiprand_uniform(rand_state)) - Vector2D(0.5f, 0.5f);
  Vector3D d = light->position + sample.x * light->dim_x + sample.y * light->dim_y - p;
  double cosTheta = dot(d, light->direction);
  double sqDist = d.norm2();
  double dist = sqrt(sqDist);
  *wi = d / dist;
  *distToLight = dist;
  *pdf = sqDist / (light->area * fabs(cosTheta));
  return cosTheta < 0 ? light->radiance : Vector3D();
}

DEVICE __inline__ Vector3D PathTracer::p_sample_L(const CudaLight light, const Vector3D p,
                             Vector3D* wi, double* distToLight,
                             double* pdf, hiprandState *rand_state) {
  switch (light.type) {
    case CudaLightType_Directional:
      return sample_L(&light_data->directional_lights[light.idx], p ,wi, distToLight, pdf);
    case CudaLightType_Point:
      return sample_L(&light_data->point_lights[light.idx], p, wi, distToLight, pdf);
    case CudaLightType_Area:
      return sample_L(&light_data->area_lights[light.idx], p, wi, distToLight, pdf, rand_state);
    default:
      return Vector3D(0, 0, 0);
  }
  return Vector3D();
}

DEVICE __inline__ Vector3D PathTracer::p_sample_f(CudaBSDF bsdf, const Vector3D wo, Vector3D *wi, double* pdf, hiprandState *rand_state) {
  switch (bsdf.type) {
      case CudaBSDFType_Diffuse:
          return sample_f(&bvh->diffuse_bsdfs[bsdf.idx], wo, wi, pdf, rand_state);
      case CudaBSDFType_Emission:
          return sample_f(&bvh->emission_bsdfs[bsdf.idx], wo, wi, pdf, rand_state);
      default:
          return Vector3D(0, 0, 0);
  }
}

DEVICE Vector3D PathTracer::estimate_direct_lighting_importance(Ray &r,
                                                const CudaIntersection &isect) {
  // Estimate the lighting from this intersection coming directly from a light.
  // To implement importance sampling, sample only from lights, not uniformly in
  // a hemisphere.

  // make a coordinate system for a hit point
  // with N aligned with the Z direction.
  Matrix3x3 o2w;

  make_coord_space(o2w, isect.n);
  Matrix3x3 w2o = o2w.T();

  // w_out points towards the source of the ray (e.g.,
  // toward the camera if this is a primary ray)
  const Vector3D hit_p = r.o + r.d * isect.t;
  const Vector3D w_out = w2o * (-r.d);
  Vector3D L_out = Vector3D(0, 0, 0);
  CudaIntersection light_isect;
  Vector3D wi;
  double distToLight, pdf;

  //NOTE: wi here is in worldpsace, unlike in the previous function
  uint16_t sample_count = 0;

  uint16_t x = blockIdx.x * blockDim.x + threadIdx.x;
  uint16_t y = blockIdx.y * blockDim.y + threadIdx.y;
  for (uint16_t i = 0; i < num_lights; i++) {
    CudaLight light = lights[i];
    int num_samples = light_data->is_delta_light(light) ? 1 : ns_area_light;
    sample_count += num_samples;
    for (int j = 0; j < num_samples; j++) {
      Vector3D radiance = p_sample_L(light, hit_p, &wi, &distToLight, &pdf, &rand_states[x + y*sampleBuffer.w]);
      Vector3D wi_o = w2o * wi;
      if (wi_o.z < 0 || radiance == 0) continue;
      Ray shadow_ray = Ray(hit_p, wi);
      shadow_ray.min_t = EPS_D;
      shadow_ray.max_t = distToLight;
      if (!bvh->intersect(shadow_ray, &light_isect)) {
        L_out += bvh->f(isect.bsdf, w_out, wi_o) * radiance * abs_cos_theta(wi_o) / pdf;
      }
    }
  }

  L_out /= sample_count;
  return L_out;
}

#define RRT 0.7

DEVICE Vector3D PathTracer::at_least_one_bounce_radiance(Ray &r, const CudaIntersection &isect) {
  Vector3D final_radiance = Vector3D();
  Vector3D throughput = Vector3D(1.0);
  
  Ray current_ray = r;
  CudaIntersection current_isect = isect;
  
  uint16_t x = blockIdx.x * blockDim.x + threadIdx.x;
  uint16_t y = blockIdx.y * blockDim.y + threadIdx.y;
  size_t idx = x + y * sampleBuffer.w;

  for (int depth = r.depth; depth <= max_ray_depth; depth++) {
    Matrix3x3 o2w;
    make_coord_space(o2w, current_isect.n);
    Matrix3x3 w2o = o2w.T();
    
    Vector3D hit_p = current_ray.o + current_ray.d * current_isect.t;
    Vector3D w_out = w2o * (-current_ray.d);
    
    // Direct lighting
    Vector3D L_direct = estimate_direct_lighting_importance(current_ray, current_isect);
    final_radiance += throughput * L_direct;
    
    bool ignore_RRT = (depth == 1 && max_ray_depth > 1);
    bool survive = ignore_RRT || (depth < max_ray_depth && hiprand_uniform(&rand_states[idx]) < RRT);
    
    if (!survive)
      break;

    // Sample new direction
    Vector3D wi;
    double pdf;
    Vector3D f = p_sample_f(current_isect.bsdf, w_out, &wi, &pdf, &rand_states[idx]);
    if (pdf == 0.0) break;

    throughput = throughput * f * abs_cos_theta(wi) / pdf / RRT;

    Ray bounce_ray = Ray(hit_p, o2w * wi);
    bounce_ray.min_t = EPS_D;
    bounce_ray.depth = depth + 1;

    CudaIntersection bounce_isect;
    if (!bvh->intersect(bounce_ray, &bounce_isect))
      break;

    current_ray = bounce_ray;
    current_isect = bounce_isect;
  }

  return final_radiance;
}


DEVICE Vector3D PathTracer::est_radiance_global_illumination(Ray &r) {
  CudaIntersection isect;
  Vector3D L_out;

  // You will extend this in assignment 3-2.
  // If no intersection occurs, we simply return black.
  // This changes if you implement hemispherical lighting for extra credit.

  // The following line of code returns a debug color depending
  // on whether ray intersection with triangles or spheres has
  // been implemented.
  //
  // REMOVE THIS LINE when you are ready to begin Part 3.
  if (!bvh->intersect(r, &isect))
    return L_out;


  if (max_ray_depth == 0)
    L_out = bvh->get_emission(isect.bsdf);
  else
    L_out = bvh->get_emission(isect.bsdf) + at_least_one_bounce_radiance(r, isect);

  // TODO (Part 3): Return the direct illumination.
  // TODO (Part 4): Accumulate the "direct" and "indirect"
  // parts of global illumination into L_out rather than just direct

  return L_out;
}

DEVICE void PathTracer::raytrace_pixel(uint16_t x, uint16_t y) {

  size_t idx = x + y * sampleBuffer.w;
  hiprand_init(1234, idx, 0, &rand_states[idx]);
  int num_samples = ns_aa;          // total samples to evaluate
  Vector2D origin = Vector2D(x, y); // bottom left corner of the pixel
  Vector3D pixel_sum = Vector3D(0, 0, 0);

  int i = 0;
  for (; i < num_samples; i++) {
    Vector2D sample = origin + Vector2D((1 - hiprand_uniform(&rand_states[idx])), (1 - hiprand_uniform(&rand_states[idx])));
    Ray r = camera.generate_ray(sample.x / sampleBuffer.w, sample.y / sampleBuffer.h);
    r.depth = 1;
    Vector3D radiance = est_radiance_global_illumination(r);
    pixel_sum += radiance;
  }
  pixel_sum /= (double)i;

  sampleBuffer.update_pixel(pixel_sum, x, y);
}
}